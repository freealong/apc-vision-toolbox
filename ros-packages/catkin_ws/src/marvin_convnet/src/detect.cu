// ---------------------------------------------------------
// Copyright (c) 2016, Andy Zeng
// 
// This file is part of the APC Vision Toolbox and is available 
// under the terms of the Simplified BSD License provided in 
// LICENSE. Please retain this notice and LICENSE if you use 
// this file (or any portion of it) in your project.
// ---------------------------------------------------------

#include "depth_utils.h"
#include "ros/ros.h"
#include "marvin_convnet/DetectObjects.h"
#include "realsense_camera/StreamSensor.h"
#include <opencv2/opencv.hpp>

// Marvin
#define DATATYPE 0
#include "marvin.hpp"

std::string shelf_net_arch_filename = "/home/yongqi/catkin_ws/src/marvin_convnet/models/competition/net.json";
std::string tote_net_arch_filename = "/home/yongqi/catkin_ws/src/marvin_convnet/models/competition/net.json";
std::string shelf_net_weights_filename = "/home/yongqi/catkin_ws/src/marvin_convnet/models/competition/weights_shelf.marvin";
std::string tote_net_weights_filename = "/home/yongqi/catkin_ws/src/marvin_convnet/models/competition/weights_tote.marvin";

// Service modes and names
std::string service_name;

// Directory to read/write all RGB-D files and response maps
std::string read_directory;

// Global buffers for sensor data retrieval
int frame_width = 640;
int frame_height = 480;
uint8_t * color_buffer = new uint8_t[frame_width * frame_height * 3];
uint8_t * HHA_buffer = new uint8_t[frame_width * frame_height * 3];

// Load Marvin FCN network architectures
marvin::Net shelf_net(shelf_net_arch_filename);
marvin::Net tote_net(tote_net_arch_filename);

// Marvin responses
StorageT* color_data_CPU = NULL;
StorageT* HHA_data_CPU = NULL;
StorageT* prob_CPU_StorageT = NULL;
ComputeT* prob_CPU_ComputeT = NULL;

ros::ServiceClient client_sensor;

const int num_apc_objects = 39;

std::string shelf_bin_ids = "ABCDEFGHIJKL";

// Service call
bool srv_detect(marvin_convnet::DetectObjects::Request  &req,
                marvin_convnet::DetectObjects::Response &res) {
  ROS_INFO("Recieved service request.");

  int bin_id = req.BinId;
  int frame_id = req.FrameId;
  res.FrameId = frame_id;

  // Get frame filenames
  std::ostringstream frame_prefix;
  frame_prefix << std::setw(6) << std::setfill('0') << frame_id;
  std::string color_frame_filename = "/frame-" + frame_prefix.str() + ".color.png";
  std::string depth_frame_filename = "/frame-" + frame_prefix.str() + ".depth.png";
  std::string raw_depth_frame_filename = "/raw/frame-" + frame_prefix.str() + ".depth.png";
  std::string HHA_frame_filename = "/HHA/frame-" + frame_prefix.str() + ".HHA.png";

  // Read color frame from disk
  cv::Mat color_frame = cv::imread(read_directory + color_frame_filename, CV_LOAD_IMAGE_COLOR);
  color_buffer = color_frame.data;
  cv::Mat HHA_frame = cv::imread(read_directory + HHA_frame_filename.c_str(), CV_LOAD_IMAGE_COLOR);
  HHA_buffer = HHA_frame.data;

  // Color: BGR format, mean subtracted
  for (int r = 0; r < frame_height; ++r)
    for (int c = 0; c < frame_width; ++c) {
      color_data_CPU[0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[0 + 3 * (c + frame_width * r)]) - ComputeT(102.9801f)); // B
      color_data_CPU[1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[1 + 3 * (c + frame_width * r)]) - ComputeT(115.9465f)); // G
      color_data_CPU[2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[2 + 3 * (c + frame_width * r)]) - ComputeT(122.7717f)); // R
      HHA_data_CPU[0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(HHA_buffer[0 + 3 * (c + frame_width * r)]) - ComputeT(102.9801f)); // B
      HHA_data_CPU[1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(HHA_buffer[1 + 3 * (c + frame_width * r)]) - ComputeT(115.9465f)); // G
      HHA_data_CPU[2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(HHA_buffer[2 + 3 * (c + frame_width * r)]) - ComputeT(122.7717f)); // R
    } 

  // Run forward pass through marvin FCN
  ROS_INFO("Forward Marvin to get segmentation results.");
  marvin::Response * rDataRGB;
  marvin::Response * rDataHHA;
  marvin::Response * rProb;
  if (bin_id == -1) {
    rDataRGB = tote_net.getResponse("data_RGB");
    rDataHHA = tote_net.getResponse("data_HHA");
    rProb = tote_net.getResponse("prob");
  } else {
    rDataRGB = shelf_net.getResponse("data_RGB");
    rDataHHA = shelf_net.getResponse("data_HHA");
    rProb = shelf_net.getResponse("prob");
  }
  hipMemcpy(rDataRGB->dataGPU, color_data_CPU, rDataRGB->numBytes(), hipMemcpyHostToDevice);
  hipMemcpy(rDataHHA->dataGPU, HHA_data_CPU, rDataHHA->numBytes(), hipMemcpyHostToDevice);
  if (bin_id == -1)
    tote_net.forward();
  else
    shelf_net.forward();
  hipMemcpy(prob_CPU_StorageT, rProb->dataGPU, rProb->numBytes(), hipMemcpyDeviceToHost);
  for (int i = 0; i < frame_height * frame_width * (num_apc_objects + 1); ++i)
    prob_CPU_ComputeT[i] = CPUStorage2ComputeT(prob_CPU_StorageT[i]);

  // Get full object list
  std::vector<std::string> all_object_names = {"background", "barkely_hide_bones", "cherokee_easy_tee_shirt", "clorox_utility_brush", "cloud_b_plush_bear", "command_hooks", "cool_shot_glue_sticks", "crayola_24_ct", "creativity_chenille_stems", "dasani_water_bottle",
                                               "dove_beauty_bar", "dr_browns_bottle_brush", "easter_turtle_sippy_cup", "elmers_washable_no_run_school_glue", "expo_dry_erase_board_eraser", "fiskars_scissors_red", "fitness_gear_3lb_dumbbell", "folgers_classic_roast_coffee", "hanes_tube_socks", "i_am_a_bunny_book",
                                               "jane_eyre_dvd", "kleenex_paper_towels", "kleenex_tissue_box", "kyjen_squeakin_eggs_plush_puppies", "laugh_out_loud_joke_book", "oral_b_toothbrush_green", "oral_b_toothbrush_red", "peva_shower_curtain_liner", "platinum_pets_dog_bowl", "rawlings_baseball",
                                               "rolodex_jumbo_pencil_cup", "safety_first_outlet_plugs", "scotch_bubble_mailer", "scotch_duct_tape", "soft_white_lightbulb", "staples_index_cards", "ticonderoga_12_pencils", "up_glucose_bottle", "womens_knit_gloves", "woods_extension_cord"};
  std::vector<std::string> selected_object_names = req.ObjectNames;

  // Remove duplicates in selected object list
  std::sort(selected_object_names.begin(), selected_object_names.end());
  selected_object_names.erase(std::unique(selected_object_names.begin(), selected_object_names.end()), selected_object_names.end());

  // Loop through each object in selected list
  for (int selected_idx = 0; selected_idx < selected_object_names.size(); selected_idx++) {
    std::string curr_object_name = selected_object_names[selected_idx];
    int curr_object_idx = std::distance(all_object_names.begin(), find(all_object_names.begin(), all_object_names.end(), curr_object_name));
    std::vector<ComputeT> predMap_object(prob_CPU_ComputeT + curr_object_idx * frame_height * frame_width, prob_CPU_ComputeT + (curr_object_idx + 1) * frame_height * frame_width);

    // Create a folder to save results
    std::ifstream file(read_directory + "/masks");
    if (file.fail())
      system(std::string("mkdir -p " + read_directory + "/masks").c_str());
    // Write segmentation response maps to 16-bit grayscale png image
    std::string result_filename = read_directory + "/masks/frame-" + frame_prefix.str() + "." + all_object_names[curr_object_idx] + ".mask.png";
    cv::Mat result_mat(frame_height, frame_width, CV_16UC1);
    for (size_t y = 0; y < frame_height; y++)
      for (size_t x = 0; x < frame_width; x++) {
        unsigned short depth_short = (unsigned short)(predMap_object[y * frame_width + x] * 65535);
        result_mat.at<unsigned short>(y, x) = depth_short;
      }
    std::vector<int> compression_params;
    compression_params.push_back(CV_IMWRITE_PNG_COMPRESSION);
    compression_params.push_back(9);
    cv::imwrite(result_filename, result_mat, compression_params);
  }

  return true;
}

int main(int argc, char **argv) {

  // Setup ROS
  ros::init(argc, argv, "marvin_convnet", ros::init_options::AnonymousName);
  ros::NodeHandle n;
  ros::NodeHandle priv_nh("~");

  // Get service parameters
  priv_nh.param("service_name", service_name, std::string("marvin_convnet"));
  priv_nh.param("read_directory", read_directory, std::string(""));

  // Assert parameters and create folder to save segmentation masks
  assert(!read_directory.empty());
  system(std::string("mkdir -p " + read_directory).c_str());

  // Start service
  ros::ServiceServer service_detect = n.advertiseService(service_name, srv_detect);

  // Connect to Realsense camera
  ROS_INFO("Reading data from directory: %s", read_directory.c_str());

  // Setup Marvin
  ROS_INFO("Loading Marvin.");
  shelf_net.Malloc(marvin::Testing);
  tote_net.Malloc(marvin::Testing);
  shelf_net.loadWeights(shelf_net_weights_filename);
  tote_net.loadWeights(tote_net_weights_filename);
  color_data_CPU = new StorageT[frame_width * frame_height * 3];
  HHA_data_CPU = new StorageT[frame_width * frame_height * 3];
  prob_CPU_StorageT = new StorageT[frame_width * frame_height * (num_apc_objects + 1)];
  prob_CPU_ComputeT = new ComputeT[frame_height * frame_width * (num_apc_objects + 1)];

  ROS_INFO("Ready.");
  ros::spin();

  return 0;
}

